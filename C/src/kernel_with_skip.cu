#include "hip/hip_runtime.h"
/*
    Konstantinos Chatziantoniou

    This is the kernel for the more complicated implementation. The kernel searches for the nearest
    neighbour of a query in a particular block. The kernel is called multiple times to
    search all adjacent blocks. A warp can SKIP the search of a block if all the queries have a nearest
    neighbour with distance smaller than the distancee from bounds.

*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include "../headers/kernel_with_skip.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define TRUE 1
#define FALSE 0
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}


__global__ 
void gpu_grid_knn_skip(float* points, float* queries, 
    int* intgr_points_per_block, int* intgr_queries_per_block, 
    int* points_per_block, int* queries_per_block, 
    float* distsances, int* neighbours,
    int num_of_points, int num_of_queries, int dimensions, int grid_d,
    int offx, int offy, int offz)
{

    extern __shared__ float shared_array[];
    // Check if the block is inbounds
    if( (int)blockIdx.x + offx  < 0 || offx + (int)blockIdx.x >= grid_d || 
    (int)blockIdx.y + offy  < 0 || offy + (int)blockIdx.y >= grid_d ||
    (int)blockIdx.z + offz  < 0 || offz + (int)blockIdx.z >= grid_d) return;

    // Block of queries
    int q_bid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    // Block of points to search
    int p_bid = blockIdx.x+offx + (blockIdx.y+offy)*gridDim.x + (blockIdx.z+offz)*gridDim.x*gridDim.y;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int stride = blockDim.x*blockDim.y;
         // For communicating the skip result between warps
    float* sh_queries = shared_array;
    float* sh_points = &shared_array[stride*dimensions];
    int* skip_wrap_res = (int*)&shared_array[stride*dimensions*2];   

    // The 0th threads of the wrap reads one element from global memory
    // and broadcasts it to the rest.
   
    int start_points = intgr_points_per_block[p_bid];
    int start_queries = intgr_queries_per_block[q_bid];
    int total_points = points_per_block[p_bid];
    int total_queries = queries_per_block[q_bid];
    float distance;
    int neighbour;

    int local_skip = TRUE;      // Init with true for threads that are out of total queries limit
    int global_skip = TRUE;
    for(int q = 0; q < total_queries; q += stride){
        // Read queries to shared memory
        int q_index = q + tid + start_queries;
        if(tid + q < total_queries){
            for(int d = 0; d < dimensions; d++){
                sh_queries[tid + d*stride] = queries[q_index + d*num_of_queries];
            }
            distance = distsances[q_index];
            neighbour = neighbours[q_index];
            // Calculate distance from boundary
            float dx = (offx==1) ? (1/(float)grid_d - fmodf(sh_queries[tid + 0*stride], 1/(float)grid_d)) : 0;
            dx = (offx == -1) ? fmodf(sh_queries[tid + 0*stride], 1/(float)grid_d) : dx;
            
            float dy = (offy==1) ? (1/(float)grid_d - fmodf(sh_queries[tid + 1*stride], 1/(float)grid_d)) : 0;
            dy = (offy == -1) ? fmodf(sh_queries[tid + 1*stride], 1/(float)grid_d) : dy;

            float dz = (offz==1) ? (1/(float)grid_d - fmodf(sh_queries[tid + 2*stride], 1/(float)grid_d)) : 0;
            dz = (offz == -1) ? fmodf(sh_queries[tid + 2*stride], 1/(float)grid_d) : dz;

            float bounds_distance = sqrtf(powf(dx,2) + powf(dy,2) + powf(dz,2));

            local_skip = (bounds_distance >= distance);
        }
        // Wrap vote
        local_skip = __all_sync(0xffffffff, local_skip);
        // Save local result to shared memory for extra-wrap comms
        if(threadIdx.x == 0){
            skip_wrap_res[threadIdx.y] = local_skip;
        }
        __syncthreads();
        if(threadIdx.x == 0){
            for(int w = 0; w < blockDim.y; w++){
                global_skip = global_skip && skip_wrap_res[w];
            }
        }
        global_skip = __shfl_sync(0xffffffff, global_skip, 0);

        if(global_skip) continue;

        for(int p = 0; p < total_points; p+= stride){
            __syncthreads();
            // Read points to shared memory
            if(p + tid < total_points){
                for(int d = 0; d < dimensions; d++){
                    sh_points[tid + d*stride] = points[start_points + p + tid + d*num_of_points];
                }
            }
            __syncthreads();

            // For each point read, calculate distance and save the minimum.
            if(local_skip) continue;

            int bounds = stride < total_points-p ? stride : total_points-p;
            if(tid + q < total_queries){
                for(int i = 0; i < bounds; i++){
                    float tempdist = 0;
                    for(int d = 0; d < dimensions; d++){
                        float tempquery = sh_queries[tid + d*stride];
                        tempdist += powf(tempquery -  sh_points[(i+tid)%bounds + d*stride], 2);
                    }
                    tempdist = sqrtf(tempdist);
                    if(tempdist < distance){
                        neighbour = start_points + p +(i+tid)%bounds;
                        distance = tempdist;
                    }
                }
            }
        }
        // Save result to global memory
        if(tid + q < total_queries){
            distsances[q_index] = distance;
            neighbours[q_index] = neighbour;
        }
    }
}


void GridKNNskip(int number_of_points, int number_of_queries, int grid_d, int dimensions,
        float* ordered_ref_points, float* ordered_queries, 
        int* intg_points_per_block, int* intg_queries_per_block,
        int* points_per_block, int* queries_per_block,
        float** out_distances, int** out_neighbours)
{
    float* distances = (float*)malloc(number_of_queries*sizeof(float));
    int* neighbours = (int*)malloc(number_of_queries*sizeof(int));

    float* dev_points; 
    float* dev_queries; 
    int *dev_intg_points_per_block, *dev_points_per_block;
    int *dev_intg_queries_per_block, *dev_queries_per_block;
    float *dev_distances;
    int* dev_neighbours;

    gpuErrchk(hipMalloc((void**)&dev_points, number_of_points*dimensions*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_queries, number_of_queries*dimensions*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_intg_points_per_block, pow(grid_d,dimensions)*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_intg_queries_per_block, pow(grid_d,dimensions)*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_points_per_block, pow(grid_d,dimensions)*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_queries_per_block, pow(grid_d,dimensions)*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_distances, number_of_queries*sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_neighbours, number_of_queries*sizeof(int)));

    gpuErrchk(hipMemcpy(dev_points, ordered_ref_points, number_of_points*dimensions*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_queries, ordered_queries, number_of_queries*dimensions*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_intg_points_per_block, intg_points_per_block, pow(grid_d,dimensions)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_intg_queries_per_block, intg_queries_per_block, pow(grid_d,dimensions)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_points_per_block, points_per_block, pow(grid_d,dimensions)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_queries_per_block, queries_per_block, pow(grid_d,dimensions)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_distances, 100, number_of_queries*sizeof(float)))
    
    int thread_groups = 2;
    dim3 blocks = {(unsigned int)grid_d, (unsigned int)grid_d, (unsigned int)grid_d};
    dim3 threads = {(unsigned int)32,(unsigned int)thread_groups,(unsigned int)1};
    uint64_t shmem = (2*thread_groups*32*dimensions)*sizeof(float) + (thread_groups)*sizeof(int);
    gpu_grid_knn_skip<<<blocks, threads, shmem>>>(dev_points, dev_queries,
                dev_intg_points_per_block, dev_intg_queries_per_block,
                dev_points_per_block, dev_queries_per_block,
                dev_distances, dev_neighbours,
                number_of_points, number_of_queries, dimensions, grid_d, 0,0,0);

    for(int x = -1; x < 2; x++){
        for(int y = -1; y < 2; y++){
            for(int z = -1; z < 2; z++){
                if(x == 0 && y == 0 && z == 0) continue;
                  gpu_grid_knn_skip<<<blocks, threads, shmem>>>(dev_points, dev_queries,
                    dev_intg_points_per_block, dev_intg_queries_per_block,
                    dev_points_per_block, dev_queries_per_block,
                    dev_distances, dev_neighbours,
                    number_of_points, number_of_queries, dimensions, grid_d, x,y,z);
            }
        }
    }
    gpuErrchk(hipMemcpy(distances, dev_distances, number_of_points*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(neighbours, dev_neighbours, number_of_points*sizeof(int), hipMemcpyDeviceToHost));
    hipFree(dev_points);
    hipFree(dev_queries);
    hipFree(dev_points_per_block);
    hipFree(dev_queries_per_block);
    hipFree(dev_intg_points_per_block);
    hipFree(dev_intg_queries_per_block);
    hipFree(dev_distances);
    hipFree(dev_neighbours);

    *out_distances = distances;
    *out_neighbours = neighbours;
}